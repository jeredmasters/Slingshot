#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>  
#include <stdio.h>

#include <time.h>
#include <assert.h>
#include <math.h>       /* pow */

#define SIZE 1024
#define FLOOR 1000
#define WALL 1000

struct Chromosome {
public:
	short *dna;
};

struct Vector2 {	
public:
	double x;
	double y;
	double length;
	__device__ double Length() {
		return pow(pow(x, 2) + pow(y, 2), 0.5);
	}
};


struct Node {
public: 
	Vector2 position;
	Vector2 velocity;
	int weight;

	
};

struct Muscle {
public:
	int strength;
	Node* nodeA;
	Node* nodeB;

};

struct Animal {
public:
	Node* nodes;
	Muscle* muscles;

};


__device__ int Rand(int min, int max, hiprandState state) {
	

	int delta = max - min;

	return hiprand_uniform(&state)*delta+min;
}

__global__ void VectorAdd(Vector2 *a, Vector2 *b, Vector2 *c, int n)
{
    int i = threadIdx.x;
	if (i < n) {
		c[i].x = a[i].length * b[i].length;
	}
}

__global__ void ApplyWalls(Node *a)
{
	int i = threadIdx.x;
	if (a[i].position.x <= 0) {
		a[i].position.x = 1;
		a[i].velocity.x = 0;
	}
	if (a[i].position.x > WALL) {
		a[i].position.x = WALL;
		a[i].velocity.x = 0;
	}
	if (a[i].position.y <= 0) {
		a[i].position.y = 1;
		a[i].velocity.y = 0;
	}
	if (a[i].position.y > FLOOR) {
		a[i].position.y = FLOOR;
		if (a[i].velocity.y > 0) {
			a[i].velocity.y = 0;
		}
	}
}

__global__ void ApplyGravity(Node *a)
{
	int i = threadIdx.x;
	a[i].velocity.y += (double)5 / (double)a[i].weight;
}

__global__ void ApplyMomentum(Node *a) 
{
	int i = threadIdx.x;
	a[i].position.x += a[i].velocity.x / 100;
	a[i].position.y += a[i].velocity.y / 100;
}

__global__ void NodeInit(Node *a)
{
	int i = threadIdx.x;
	a[i].weight = 20;
}

__global__ void AnimalInit(Animal *a) {
	int i = threadIdx.x;
}

__global__ void ChromosomeInit(Chromosome *c, short* o, int length) {
	int i = threadIdx.x;
	hiprandState state;
	hiprand_init(clock(), i, 0, &state);
	c[i].dna = new short[length];
	for (int j = 0; j < length; j++) {
		c[i].dna[j] = Rand(1, 200, state);
		o[j] = c[i].dna[j];
	}
}

int main()
{
	short *output = (short *)malloc(SIZE * sizeof(short));
	Node *d_a, *d_b;
	Chromosome *d_c;
	short * d_o;

	hipSetDevice(0);

	//a = (int *)malloc(SIZE * sizeof(int));
	//b = (int *)malloc(SIZE * sizeof(int));
	//output = (int *)malloc(SIZE * sizeof(int));

	hipMalloc(&d_a, SIZE * sizeof(Node));
	hipMalloc(&d_c, SIZE * sizeof(Chromosome));
	hipMalloc(&d_o, SIZE * sizeof(short));

	//hipMemcpy(d_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(d_b, b, SIZE * sizeof(int), hipMemcpyHostToDevice);
	//hipMemcpy(d_c, c, SIZE * sizeof(int), hipMemcpyHostToDevice);

	NodeInit <<< 1, SIZE >>> (d_a);

	for (int click = 0; click < 100; click++) {
		NodeInit << < 1, SIZE >> > (d_a);
		//ApplyGravity << < 1, SIZE >> > (d_a);
		//ApplyMomentum << < 1, SIZE >> > (d_a);
		//ApplyWalls << < 1, SIZE >> > (d_a);

		hipError_t cudaStatus = hipMemcpy(output, d_o, SIZE * sizeof(short), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		else {
			//for (int i = 0; i < 1; i++) {
				int t = 80;
				printf("%d: %d\n", click, output[t]);
			//}
		}
	}



	//free(a);
	//free(b);
	free(output);

	hipFree(d_a);
	//hipFree(d_b);
	//hipFree(d_c);

	getchar();

}